#include "hip/hip_runtime.h"
/*
 * CudaAcceleratedAlgorithm.cpp
 * Author: Rémi PIOTAIX <remi.piotaix@gmail.com>
 * 
 * Created on: 21 mai 2013     
 */

#include "Object.h"


#include "Instance.h"

template<typename P, typename W>
CudaAcceleratedAlgorithm<P, W>::CudaAcceleratedAlgorithm() :
AbstractKnapsackAlgorithm<P, W>() {

}

template<typename P, typename W>
CudaAcceleratedAlgorithm<P, W>::CudaAcceleratedAlgorithm(const CudaAcceleratedAlgorithm<P, W>& orig) :
AbstractKnapsackAlgorithm<P, W>(orig) {

}

template<typename P, typename W>
CudaAcceleratedAlgorithm<P, W>::~CudaAcceleratedAlgorithm() {

}

template<typename P, typename W, unsigned repeat>
__global__ void kernel(P* data, P* retour, W objSize, P objUtility, unsigned size, unsigned decalBlock = 0) throw () {
    unsigned i = blockDim.x * (blockIdx.x + decalBlock) + threadIdx.x;
    unsigned sep = blockDim.x * gridDim.x;
    P retValue;

#pragma unroll
    for (unsigned num = 0; num < repeat; num++) {
        if (i < size) {
            retValue = data[i];
            if (i >= objSize)
                retValue = max(retValue, data[i - objSize] + objUtility);

            retour[i] = retValue;
        }

        i += sep;
    }

}

template<typename T>
void fillWith(T* tab, T value, int tabSize) {
    for (int i = 0; i < tabSize; i++) {
        tab[i] = value;
    }
}

std::ostream& operator<<(std::ostream& os, const hipError_t & err) {
    os << hipGetErrorString(err);

    return os;
}

template<typename T>
void printTab(T* tab, size_t taille) {
    using namespace std;

    for (size_t i = 0; i < taille; i++) {
        cout << setw(5) << tab[i];
    }
    cout << endl;
}

void HANDLE(hipError_t err) {
    using namespace std;

    if (err != hipSuccess)
        cout << err << endl;
}

template<typename P, typename W >
P CudaAcceleratedAlgorithm<P, W>::run(const Instance<P, W>& instance) {
    using namespace std;

    int nbObjects = instance.getNbObjects();

    hipDeviceProp_t prop;
    HANDLE(hipSetDevice(0));
    HANDLE(hipGetDeviceProperties(&prop, 0));
    int block_max_size = prop.maxGridSize[0];

    size_t dataSize = instance.getKnapsackSize() + 1;

    P* d_retour;
    P* d_data;

    int threadsPerBlock = 1 << min((int) log2((float) dataSize), 8);
    double blocks = (double) dataSize / threadsPerBlock / 8;
    int blocksPerGrid = (int) blocks + 1;

    HANDLE(hipMalloc(&d_retour, sizeof (P) * dataSize));
    HANDLE(hipMalloc(&d_data, sizeof (P) * dataSize));

    HANDLE(hipMemset(d_data, 0, sizeof (P) * dataSize));
    HANDLE(hipMemset(d_retour, 0, sizeof (P) * dataSize));

    unsigned i = 1;
    for (typename vector<Object<P, W> >::const_iterator iterator = instance.getObjects().begin();
            iterator != instance.getObjects().end(); iterator++) {
        //cout << "For BEGIN " << i << endl;

        const Object<P, W>& o = *iterator;

        unsigned decalBlock = 0;
        while (blocksPerGrid - decalBlock >= block_max_size) {
            kernel<P, W, 8> << <block_max_size, threadsPerBlock >> >(d_data, d_retour, o.getWeight(), o.getUtility(), dataSize, decalBlock);
            decalBlock += block_max_size;
        }
        if (blocksPerGrid > decalBlock)
            kernel<P, W, 8> << <blocksPerGrid - decalBlock, threadsPerBlock >> >(d_data, d_retour, o.getWeight(), o.getUtility(), dataSize, decalBlock);

        P* temp = d_data;
        d_data = d_retour;
        d_retour = temp;

        cout << setw(10) << i << " /" << setw(10) << nbObjects << setw(10) << dataSize << '\r' << flush;
        i++;
    }
    P resultat;
    HANDLE(hipMemcpy(&resultat, d_data + dataSize - 1, sizeof (P), hipMemcpyDeviceToHost));

    HANDLE(hipFree(d_data));
    HANDLE(hipFree(d_retour));
    cout << endl;


    return resultat;
}
