
#include <hip/hip_runtime.h>



template<typename P, typename W>
__global__ void kernel(P* data, P* retour, W objSize, P objUtility, W knapsackSize, W wmax) {
    unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
    retour[wmax + i] = max(data[wmax + i], data[wmax + i - objSize] + objUtility);
}

template<typename P, typename W>
void cudaAcceleratedAlgorithm(dim3 blocksPerGrid, dim3 threadsPerBlock, P* data, P* retour, W objSize, P objUtility, W knapsackSize, W wmax) {
    kernel<P, W><<<blocksPerGrid, threadsPerBlock >>>(data, retour, objSize, objUtility, knapsackSize, wmax);
}